#include "hip/hip_runtime.h"
/* Copyright 2024 The JAX Authors.

Licensed under the Apache License, Version 2.0 (the "License");
you may not use this file except in compliance with the License.
You may obtain a copy of the License at

    http://www.apache.org/licenses/LICENSE-2.0

Unless required by applicable law or agreed to in writing, software
distributed under the License is distributed on an "AS IS" BASIS,
WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
See the License for the specific language governing permissions and
limitations under the License.
==============================================================================*/

#include "kernel_helpers.h"
#include "kernels.h"
#include "stdio.h"
#include <hip/hip_bf16.h>
#include <hip/hip_fp16.h>
#include <iostream>

namespace {

#define DISPATCH_DOUBLE_FLOAT_HALF_AND_BFLOAT_INOUT_TYPES(TYPEIN, TYPEOUT,     \
                                                          NAME, ...)           \
  switch (TYPEIN) {                                                            \
  case gpu_ops::ElementType::F64: {                                            \
    using scalar_t_in = double;                                                \
    using accscalar_t = double;                                                \
    switch (TYPEOUT) {                                                         \
    case gpu_ops::ElementType::F64: {                                          \
      using scalar_t_out = double;                                             \
      __VA_ARGS__;                                                             \
      break;                                                                   \
    }                                                                          \
    case gpu_ops::ElementType::F32: {                                          \
      using scalar_t_out = float;                                              \
      __VA_ARGS__;                                                             \
      break;                                                                   \
    }                                                                          \
    case gpu_ops::ElementType::F16: {                                          \
      using scalar_t_out = __half;                                             \
      __VA_ARGS__;                                                             \
      break;                                                                   \
    }                                                                          \
    case gpu_ops::ElementType::BF16: {                                         \
      using scalar_t_out = __hip_bfloat16;                                      \
      __VA_ARGS__;                                                             \
      break;                                                                   \
    }                                                                          \
    default:                                                                   \
      break;                                                                   \
    }                                                                          \
    break;                                                                     \
  }                                                                            \
  case gpu_ops::ElementType::F32: {                                            \
    using scalar_t_in = float;                                                 \
    using accscalar_t = float;                                                 \
    switch (TYPEOUT) {                                                         \
    case gpu_ops::ElementType::F64: {                                          \
      using scalar_t_out = double;                                             \
      __VA_ARGS__;                                                             \
      break;                                                                   \
    }                                                                          \
    case gpu_ops::ElementType::F32: {                                          \
      using scalar_t_out = float;                                              \
      __VA_ARGS__;                                                             \
      break;                                                                   \
    }                                                                          \
    case gpu_ops::ElementType::F16: {                                          \
      using scalar_t_out = __half;                                             \
      __VA_ARGS__;                                                             \
      break;                                                                   \
    }                                                                          \
    case gpu_ops::ElementType::BF16: {                                         \
      using scalar_t_out = __hip_bfloat16;                                      \
      __VA_ARGS__;                                                             \
      break;                                                                   \
    }                                                                          \
    default:                                                                   \
      break;                                                                   \
    }                                                                          \
    break;                                                                     \
  }                                                                            \
  case gpu_ops::ElementType::F16: {                                            \
    using scalar_t_in = __half;                                                \
    using accscalar_t = float;                                                 \
    switch (TYPEOUT) {                                                         \
    case gpu_ops::ElementType::F64: {                                          \
      using scalar_t_out = double;                                             \
      __VA_ARGS__;                                                             \
      break;                                                                   \
    }                                                                          \
    case gpu_ops::ElementType::F32: {                                          \
      using scalar_t_out = float;                                              \
      __VA_ARGS__;                                                             \
      break;                                                                   \
    }                                                                          \
    case gpu_ops::ElementType::F16: {                                          \
      using scalar_t_out = __half;                                             \
      __VA_ARGS__;                                                             \
      break;                                                                   \
    }                                                                          \
    case gpu_ops::ElementType::BF16: {                                         \
      using scalar_t_out = __hip_bfloat16;                                      \
      __VA_ARGS__;                                                             \
      break;                                                                   \
    }                                                                          \
    default:                                                                   \
      break;                                                                   \
    }                                                                          \
    break;                                                                     \
  }                                                                            \
  case gpu_ops::ElementType::BF16: {                                           \
    using scalar_t_in = __hip_bfloat16;                                         \
    using accscalar_t = float;                                                 \
    switch (TYPEOUT) {                                                         \
    case gpu_ops::ElementType::F64: {                                          \
      using scalar_t_out = double;                                             \
      __VA_ARGS__;                                                             \
      break;                                                                   \
    }                                                                          \
    case gpu_ops::ElementType::F32: {                                          \
      using scalar_t_out = float;                                              \
      __VA_ARGS__;                                                             \
      break;                                                                   \
    }                                                                          \
    case gpu_ops::ElementType::F16: {                                          \
      using scalar_t_out = __half;                                             \
      __VA_ARGS__;                                                             \
      break;                                                                   \
    }                                                                          \
    case gpu_ops::ElementType::BF16: {                                         \
      using scalar_t_out = __hip_bfloat16;                                      \
      __VA_ARGS__;                                                             \
      break;                                                                   \
    }                                                                          \
    default:                                                                   \
      break;                                                                   \
    }                                                                          \
    break;                                                                     \
  }                                                                            \
  default:                                                                     \
    break;                                                                     \
  }

template <typename U>
__device__ void cuWelfordOnlineSum(const U curr, U &mu, U &sigma2, U &count) {
  count = count + U(1);
  U delta = curr - mu;
  U lmean = mu + delta / count;
  mu = lmean;
  U delta2 = curr - lmean;
  sigma2 = sigma2 + delta * delta2;
}

template <typename U>
__device__ void cuChanOnlineSum(const U muB, const U sigma2B, const U countB,
                                U &mu, U &sigma2, U &count) {
  U delta = muB - mu;
  U nA = count;
  U nB = countB;
  count = count + countB;
  U nX = count;
  if (nX > U(0)) {
    nA = nA / nX;
    nB = nB / nX;
    mu = nA * mu + nB * muB;
    sigma2 = sigma2 + sigma2B + delta * delta * nA * nB * nX;
  } else {
    mu = U(0);
    sigma2 = U(0);
  }
}

template <typename U> __device__ void cuRMSOnlineSum(const U curr, U &sigma2) {
  sigma2 = sigma2 + curr * curr;
}

template <typename U>
__device__ void cuChanRMSOnlineSum(const U sigma2B, U &sigma2) {
  sigma2 = sigma2 + sigma2B;
}

template <typename T, typename U>
__device__ void cuWelfordMuSigma2(const T *__restrict__ vals, const int n1,
                                  const int n2, const int i1, U &mu, U &sigma2,
                                  U *buf, bool rms_only) {
  // Assumptions:
  // 1) blockDim.x == warpSize
  // 2) Tensor is contiguous
  // 3) 2*blockDim.y*sizeof(U)+blockDim.y*sizeof(int) shared memory available.
  //
  // compute variance and mean over n2
  U count = U(0);
  mu = U(0);
  sigma2 = U(0);
  if (i1 < n1) {
    // one warp normalizes one n1 index,
    // synchronization is implicit
    // initialize with standard Welford algorithm
    const int numx = blockDim.x * blockDim.y;
    const int thrx = threadIdx.x + threadIdx.y * blockDim.x;
    const T *lvals = vals + i1 * n2;
    int l = 4 * thrx;
    for (; l + 3 < n2; l += 4 * numx) {
      for (int k = 0; k < 4; ++k) {
        U curr = static_cast<U>(lvals[l + k]);
        if (!rms_only) {
          cuWelfordOnlineSum<U>(curr, mu, sigma2, count);
        } else {
          cuRMSOnlineSum<U>(curr, sigma2);
        }
      }
    }
    for (; l < n2; ++l) {
      U curr = static_cast<U>(lvals[l]);
      if (!rms_only) {
        cuWelfordOnlineSum<U>(curr, mu, sigma2, count);
      } else {
        cuRMSOnlineSum<U>(curr, sigma2);
      }
    }
    // intra-warp reductions
    for (int l = 0; l <= 4; ++l) {
      int srcLaneB = (threadIdx.x + (1 << l)) & 31;
      U sigma2B = __shfl_sync(0xffffffff, sigma2, srcLaneB, warpSize);
      if (!rms_only) {
        U muB = __shfl_sync(0xffffffff, mu, srcLaneB, warpSize);
        U countB = __shfl_sync(0xffffffff, count, srcLaneB, warpSize);
        cuChanOnlineSum<U>(muB, sigma2B, countB, mu, sigma2, count);
      } else {
        cuChanRMSOnlineSum<U>(sigma2B, sigma2);
      }
    }
    // threadIdx.x == 0 has correct values for each warp
    // inter-warp reductions
    if (blockDim.y > 1) {
      U *ubuf = (U *)buf;
      U *ibuf = (U *)(ubuf + blockDim.y);
      for (int offset = blockDim.y / 2; offset > 0; offset /= 2) {
        // upper half of warps write to shared
        if (threadIdx.x == 0 && threadIdx.y >= offset &&
            threadIdx.y < 2 * offset) {
          const int wrt_y = threadIdx.y - offset;
          if (!rms_only) {
            ubuf[2 * wrt_y] = mu;
            ibuf[wrt_y] = count;
          }
          ubuf[2 * wrt_y + 1] = sigma2;
        }
        __syncthreads();
        // lower half merges
        if (threadIdx.x == 0 && threadIdx.y < offset) {
          U sigma2B = ubuf[2 * threadIdx.y + 1];
          if (!rms_only) {
            U muB = ubuf[2 * threadIdx.y];
            U countB = ibuf[threadIdx.y];
            cuChanOnlineSum<U>(muB, sigma2B, countB, mu, sigma2, count);
          } else {
            cuChanRMSOnlineSum<U>(sigma2B, sigma2);
          }
        }
        __syncthreads();
      }
      // threadIdx.x = 0 && threadIdx.y == 0 only thread that has correct values
      if (threadIdx.x == 0 && threadIdx.y == 0) {
        if (!rms_only) {
          ubuf[0] = mu;
        }
        ubuf[1] = sigma2;
      }
      __syncthreads();
      if (!rms_only) {
        mu = ubuf[0];
      }
      sigma2 = ubuf[1] / U(n2);
      // don't care about final value of count, we know count == n2
    } else {
      if (!rms_only) {
        mu = __shfl_sync(0xffffffff, mu, 0, warpSize);
      }
      sigma2 = __shfl_sync(0xffffffff, sigma2 / U(n2), 0, warpSize);
    }
  }
}

template <>
__device__ void cuWelfordMuSigma2(const __half *__restrict__ vals, const int n1,
                                  const int n2, const int i1, float &mu,
                                  float &sigma2, float *buf, bool rms_only) {
  // Assumptions:
  // 1) blockDim.x == warpSize
  // 2) Tensor is contiguous
  // 3) 2*blockDim.y*sizeof(U)+blockDim.y*sizeof(int) shared memory available.
  //
  // compute variance and mean over n2
  float count = 0.0f;
  mu = float(0);
  sigma2 = float(0);
  if (i1 < n1) {
    // one warp normalizes one n1 index,
    // synchronization is implicit
    // initialize with standard Welford algorithm
    const int numx = blockDim.x * blockDim.y;
    const int thrx = threadIdx.x + threadIdx.y * blockDim.x;
    const __half *lvals = vals + i1 * n2;
    int l = 8 * thrx;
    if ((((size_t)lvals) & 3) != 0) {
      // 16 bit alignment
      // first thread consumes first point
      if (thrx == 0) {
        float curr = static_cast<float>(lvals[0]);
        if (!rms_only) {
          cuWelfordOnlineSum(curr, mu, sigma2, count);
        } else {
          cuRMSOnlineSum(curr, sigma2);
        }
      }
      ++l;
    }
    // at this point, lvals[l] are 32 bit aligned for all threads.
    for (; l + 7 < n2; l += 8 * numx) {
      for (int k = 0; k < 8; k += 2) {
        float2 curr = __half22float2(*((__half2 *)(lvals + l + k)));
        if (!rms_only) {
          cuWelfordOnlineSum(curr.x, mu, sigma2, count);
          cuWelfordOnlineSum(curr.y, mu, sigma2, count);
        } else {
          cuRMSOnlineSum(curr.x, sigma2);
          cuRMSOnlineSum(curr.y, sigma2);
        }
      }
    }
    for (; l < n2; ++l) {
      float curr = static_cast<float>(lvals[l]);
      if (!rms_only) {
        cuWelfordOnlineSum(curr, mu, sigma2, count);
      } else {
        cuRMSOnlineSum(curr, sigma2);
      }
    }
    // intra-warp reductions
    for (int l = 0; l <= 4; ++l) {
      int srcLaneB = (threadIdx.x + (1 << l)) & 31;
      float sigma2B = __shfl_sync(0xffffffff, sigma2, srcLaneB, warpSize);
      if (!rms_only) {
        float muB = __shfl_sync(0xffffffff, mu, srcLaneB, warpSize);
        float countB = __shfl_sync(0xffffffff, count, srcLaneB, warpSize);
        cuChanOnlineSum(muB, sigma2B, countB, mu, sigma2, count);
      } else {
        cuChanRMSOnlineSum(sigma2B, sigma2);
      }
    }
    // threadIdx.x == 0 has correct values for each warp
    // inter-warp reductions
    if (blockDim.y > 1) {
      float *ubuf = (float *)buf;
      float *ibuf = (float *)(ubuf + blockDim.y);
      for (int offset = blockDim.y / 2; offset > 0; offset /= 2) {
        // upper half of warps write to shared
        if (threadIdx.x == 0 && threadIdx.y >= offset &&
            threadIdx.y < 2 * offset) {
          const int wrt_y = threadIdx.y - offset;
          ubuf[2 * wrt_y + 1] = sigma2;
          if (!rms_only) {
            ubuf[2 * wrt_y] = mu;
            ibuf[wrt_y] = count;
          }
        }
        __syncthreads();
        // lower half merges
        if (threadIdx.x == 0 && threadIdx.y < offset) {
          float sigma2B = ubuf[2 * threadIdx.y + 1];
          if (!rms_only) {
            float muB = ubuf[2 * threadIdx.y];
            float countB = ibuf[threadIdx.y];
            cuChanOnlineSum(muB, sigma2B, countB, mu, sigma2, count);
          } else {
            cuChanRMSOnlineSum(sigma2B, sigma2);
          }
        }
        __syncthreads();
      }
      // threadIdx.x = 0 && threadIdx.y == 0 only thread that has correct values
      if (threadIdx.x == 0 && threadIdx.y == 0) {
        if (!rms_only) {
          ubuf[0] = mu;
        }
        ubuf[1] = sigma2;
      }
      __syncthreads();
      if (!rms_only) {
        mu = ubuf[0];
      }
      sigma2 = ubuf[1] / float(n2);
      // don't care about final value of count, we know count == n2
    } else {
      if (!rms_only) {
        mu = __shfl_sync(0xffffffff, mu, 0, warpSize);
      }
      sigma2 = __shfl_sync(0xffffffff, sigma2 / float(n2), 0, warpSize);
    }
  }
}

// This is the un-specialized struct.  Note that we prevent instantiation of
// this struct by putting an undefined symbol in the function body so it won't
// compile.
//  template <typename T>
//  struct SharedMemory
//  {
//      // Ensure that we won't compile any un-specialized types
//      __device__ T *getPointer()
//      {
//          extern __device__ void error(void);
//          error();
//          return NULL;
//      }
//  };
// https://github.com/NVIDIA/apex/issues/246
template <typename T> struct SharedMemory;

template <> struct SharedMemory<float> {
  __device__ float *getPointer() {
    extern __shared__ float s_float[];
    return s_float;
  }
};

template <> struct SharedMemory<double> {
  __device__ double *getPointer() {
    extern __shared__ double s_double[];
    return s_double;
  }
};

template <typename T, typename U, typename V>
__device__ void cuApplyLayerNorm_(V *__restrict__ output_vals,
                                  U *__restrict__ mean, U *__restrict__ invvar,
                                  const T *__restrict__ vals, const int n1,
                                  const int n2, const U epsilon,
                                  const V *__restrict__ gamma,
                                  const V *__restrict__ beta, bool rms_only) {
  // Assumptions:
  // 1) blockDim.x == warpSize
  // 2) Tensors are contiguous
  //
  for (auto i1 = blockIdx.y; i1 < n1; i1 += gridDim.y) {
    SharedMemory<U> shared;
    U *buf = shared.getPointer();
    U mu, sigma2;
    cuWelfordMuSigma2(vals, n1, n2, i1, mu, sigma2, buf, rms_only);

    const T *lvals = vals + i1 * n2;
    V *ovals = output_vals + i1 * n2;
    U c_invvar = rsqrt(sigma2 + epsilon);
    const int numx = blockDim.x * blockDim.y;
    const int thrx = threadIdx.x + threadIdx.y * blockDim.x;
    if (gamma != NULL && (beta != NULL || rms_only)) {
      for (int i = thrx; i < n2; i += numx) {
        U curr = static_cast<U>(lvals[i]);
        if (!rms_only) {
          ovals[i] =
              gamma[i] * static_cast<V>(c_invvar * (curr - mu)) + beta[i];
        } else {
          ovals[i] = gamma[i] * static_cast<V>(c_invvar * curr);
        }
      }
    } else {
      for (int i = thrx; i < n2; i += numx) {
        U curr = static_cast<U>(lvals[i]);
        if (!rms_only) {
          ovals[i] = static_cast<V>(c_invvar * (curr - mu));
        } else {
          ovals[i] = static_cast<V>(c_invvar * curr);
        }
      }
    }
    if (threadIdx.x == 0 && threadIdx.y == 0) {
      if (!rms_only) {
        mean[i1] = mu;
      }
      invvar[i1] = c_invvar;
    }
    __syncthreads();
  }
}

template <typename T, typename U, typename V = T>
__global__ void
cuApplyRMSNorm(V *__restrict__ output_vals, U *__restrict__ invvar,
               const T *__restrict__ vals, const int n1, const int n2,
               const U epsilon, const V *__restrict__ gamma) {
  cuApplyLayerNorm_<T, U, V>(output_vals, NULL, invvar, vals, n1, n2, epsilon,
                             gamma, NULL, true);
}

template <typename T, typename U, typename V = T>
void HostApplyRMSNorm(hipStream_t stream, V *output, U *invvar, const T *input,
                      int n1, int n2, double epsilon, const V *gamma) {
  auto getMaxGridY = []() {
    int device;
    int val;
    hipGetDevice(&device);
    hipDeviceGetAttribute(&val, hipDeviceAttributeMaxGridDimY, device);
    return val;
  };
  const dim3 threads(32, 4, 1);
  const uint64_t maxGridY = getMaxGridY();
  const dim3 blocks(1, std::min((uint64_t)n1, maxGridY), 1);
  int nshared =
      threads.y > 1 ? threads.y * sizeof(U) + (threads.y / 2) * sizeof(U) : 0;
  cuApplyRMSNorm<<<blocks, threads, nshared, stream>>>(
      output, invvar, input, n1, n2, U(epsilon), gamma);
}

template <typename T, typename U, typename V>
__device__ void cuLoadWriteStridedInputs(
    const int i1_block, const int thr_load_row_off, const int thr_load_col_off,
    const int i2_off, const int row_stride, U *warp_buf1, U *warp_buf2,
    const T *input, const V *dout, const int i1_end, const int n2,
    const U *__restrict__ mean, const U *__restrict__ invvar, bool rms_only) {
  int i1 = i1_block + thr_load_row_off;
  if (i1 < i1_end) {
    U curr_mean;
    if (!rms_only) {
      curr_mean = mean[i1];
    }
    U curr_invvar = invvar[i1];
    for (int k = 0; k < blockDim.y; ++k) {
      int i2 = i2_off + k;
      int load_idx = i1 * n2 + i2;
      int write_idx = thr_load_row_off * row_stride + thr_load_col_off + k;
      if (i2 < n2) {
        U curr_input = static_cast<U>(input[load_idx]);
        U curr_dout = static_cast<U>(dout[load_idx]);
        if (!rms_only) {
          warp_buf1[write_idx] = curr_dout;
          warp_buf2[write_idx] =
              curr_dout * (curr_input - curr_mean) * curr_invvar;
        } else {
          warp_buf2[write_idx] = curr_dout * (curr_input)*curr_invvar;
        }
      } else {
        if (!rms_only) {
          warp_buf1[write_idx] = U(0);
        }
        warp_buf2[write_idx] = U(0);
      }
    }
  } else {
    for (int k = 0; k < blockDim.y; ++k) {
      int write_idx = thr_load_row_off * row_stride + thr_load_col_off + k;
      if (!rms_only) {
        warp_buf1[write_idx] = U(0);
      }
      warp_buf2[write_idx] = U(0);
    }
  }
}

template <typename T, typename U, typename V>
__device__ void cuLoadAddStridedInputs(
    const int i1_block, const int thr_load_row_off, const int thr_load_col_off,
    const int i2_off, const int row_stride, U *warp_buf1, U *warp_buf2,
    const T *input, const V *dout, const int i1_end, const int n2,
    const U *__restrict__ mean, const U *__restrict__ invvar, bool rms_only) {
  int i1 = i1_block + thr_load_row_off;
  if (i1 < i1_end) {
    U curr_mean;
    if (!rms_only) {
      curr_mean = mean[i1];
    }
    U curr_invvar = invvar[i1];
    for (int k = 0; k < blockDim.y; ++k) {
      int i2 = i2_off + k;
      int load_idx = i1 * n2 + i2;
      int write_idx = thr_load_row_off * row_stride + thr_load_col_off + k;
      if (i2 < n2) {
        U curr_input = static_cast<U>(input[load_idx]);
        U curr_dout = static_cast<U>(dout[load_idx]);
        if (!rms_only) {
          warp_buf1[write_idx] += curr_dout;
          warp_buf2[write_idx] +=
              curr_dout * (curr_input - curr_mean) * curr_invvar;
        } else {
          warp_buf2[write_idx] += curr_dout * (curr_input)*curr_invvar;
        }
      }
    }
  }
}

template <typename T, typename U, typename V>
__global__ void cuComputePartGradGammaBeta(
    const V *__restrict__ dout, const T *__restrict__ input, const int n1,
    const int n2, const U *__restrict__ mean, const U *__restrict__ invvar,
    U epsilon, U *part_grad_gamma, U *part_grad_beta, bool rms_only) {
  const int numsegs_n1 =
      (n1 + blockDim.y * blockDim.y - 1) / (blockDim.y * blockDim.y);
  const int segs_per_block = (numsegs_n1 + gridDim.y - 1) / gridDim.y;
  const int i1_beg = blockIdx.y * segs_per_block * blockDim.y * blockDim.y;
  const int i1_beg_plus_one =
      (blockIdx.y + 1) * segs_per_block * blockDim.y * blockDim.y;
  const int i1_end = i1_beg_plus_one < n1 ? i1_beg_plus_one : n1;
  const int row_stride = blockDim.x + 1;
  const int thr_load_col_off = (threadIdx.x * blockDim.y) & (blockDim.x - 1);
  const int thr_load_row_off =
      (threadIdx.x * blockDim.y) / blockDim.x + threadIdx.y * blockDim.y;
  const int i2_off = blockIdx.x * blockDim.x + thr_load_col_off;
  SharedMemory<U> shared;
  U *buf = shared.getPointer(); // buf has at least blockDim.x * blockDim.y *
                                // blockDim.y + (blockDim.y -
                                // 1)*(blockDim.x/blockDim.y) elements
  U *warp_buf1 = (U *)buf;
  U *warp_buf2 = warp_buf1 + blockDim.y * blockDim.y * row_stride;
  // compute partial sums from strided inputs
  // do this to increase number of loads in flight
  cuLoadWriteStridedInputs(i1_beg, thr_load_row_off, thr_load_col_off, i2_off,
                           row_stride, warp_buf1, warp_buf2, input, dout,
                           i1_end, n2, mean, invvar, rms_only);
  for (int i1_block = i1_beg + blockDim.y * blockDim.y; i1_block < i1_end;
       i1_block += blockDim.y * blockDim.y) {
    cuLoadAddStridedInputs(i1_block, thr_load_row_off, thr_load_col_off, i2_off,
                           row_stride, warp_buf1, warp_buf2, input, dout,
                           i1_end, n2, mean, invvar, rms_only);
  }
  __syncthreads();
  // inter-warp reductions
  // sum within each warp
  U acc1 = U(0);
  U acc2 = U(0);
  for (int k = 0; k < blockDim.y; ++k) {
    int row1 = threadIdx.y + k * blockDim.y;
    int idx1 = row1 * row_stride + threadIdx.x;
    if (!rms_only) {
      acc1 += warp_buf1[idx1];
    }
    acc2 += warp_buf2[idx1];
  }
  if (!rms_only) {
    warp_buf1[threadIdx.y * row_stride + threadIdx.x] = acc1;
  }
  warp_buf2[threadIdx.y * row_stride + threadIdx.x] = acc2;
  __syncthreads();
  // sum all warps
  for (int offset = blockDim.y / 2; offset > 1; offset /= 2) {
    if (threadIdx.y < offset) {
      int row1 = threadIdx.y;
      int row2 = threadIdx.y + offset;
      int idx1 = row1 * row_stride + threadIdx.x;
      int idx2 = row2 * row_stride + threadIdx.x;
      if (!rms_only) {
        warp_buf1[idx1] += warp_buf1[idx2];
      }
      warp_buf2[idx1] += warp_buf2[idx2];
    }
    __syncthreads();
  }
  int i2 = blockIdx.x * blockDim.x + threadIdx.x;
  if (threadIdx.y == 0 && i2 < n2) {
    int row1 = threadIdx.y;
    int row2 = threadIdx.y + 1;
    int idx1 = row1 * row_stride + threadIdx.x;
    int idx2 = row2 * row_stride + threadIdx.x;
    if (!rms_only) {
      part_grad_beta[blockIdx.y * n2 + i2] = warp_buf1[idx1] + warp_buf1[idx2];
    }
    part_grad_gamma[blockIdx.y * n2 + i2] = warp_buf2[idx1] + warp_buf2[idx2];
  }
}

template <typename U, typename V>
__global__ void
cuComputeGradGammaBeta(const U *part_grad_gamma, const U *part_grad_beta,
                       const int part_size, const int n1, const int n2,
                       V *grad_gamma, V *grad_beta, bool rms_only) {
  // sum partial gradients for gamma and beta
  SharedMemory<U> shared;
  U *buf = shared.getPointer();
  int i2 = blockIdx.x * blockDim.x + threadIdx.x;
  if (i2 < n2) {
    // each warp does sequential reductions until reduced part_size is num_warps
    int num_warp_reductions = part_size / blockDim.y;
    U sum_gamma = U(0);
    U sum_beta = U(0);
    const U *part_grad_gamma_ptr =
        part_grad_gamma + threadIdx.y * num_warp_reductions * n2 + i2;
    const U *part_grad_beta_ptr =
        part_grad_beta + threadIdx.y * num_warp_reductions * n2 + i2;
    for (int warp_offset = 0; warp_offset < num_warp_reductions;
         ++warp_offset) {
      sum_gamma += part_grad_gamma_ptr[warp_offset * n2];
      if (!rms_only) {
        sum_beta += part_grad_beta_ptr[warp_offset * n2];
      }
    }
    // inter-warp reductions
    const int nbsize3 = blockDim.x * blockDim.y / 2;
    for (int offset = blockDim.y / 2; offset >= 1; offset /= 2) {
      // top half write to shared memory
      if (threadIdx.y >= offset && threadIdx.y < 2 * offset) {
        const int write_idx = (threadIdx.y - offset) * blockDim.x + threadIdx.x;
        buf[write_idx] = sum_gamma;
        if (!rms_only) {
          buf[write_idx + nbsize3] = sum_beta;
        }
      }
      __syncthreads();
      // bottom half sums
      if (threadIdx.y < offset) {
        const int read_idx = threadIdx.y * blockDim.x + threadIdx.x;
        sum_gamma += buf[read_idx];
        if (!rms_only) {
          sum_beta += buf[read_idx + nbsize3];
        }
      }
      __syncthreads();
    }
    // write out fully summed gradients
    if (threadIdx.y == 0) {
      grad_gamma[i2] = sum_gamma;
      if (!rms_only) {
        grad_beta[i2] = sum_beta;
      }
    }
  }
}

template <typename T, typename U, typename V>
__global__ void
cuComputeGradInput(const V *__restrict__ dout, const T *__restrict__ input,
                   const int n1, const int n2, const U *__restrict__ mean,
                   const U *__restrict__ invvar, U epsilon, const V *gamma,
                   T *grad_input, bool rms_only) {
  for (auto i1 = blockIdx.y; i1 < n1; i1 += gridDim.y) {
    U sum_loss1 = U(0);
    U sum_loss2 = U(0);
    U c_mean;
    if (!rms_only) {
      c_mean = mean[i1];
    }
    const U c_invvar = invvar[i1];
    const T *k_input = input + i1 * n2;
    const V *k_dout = dout + i1 * n2;
    const int numx = blockDim.x * blockDim.y;
    const int thrx = threadIdx.x + threadIdx.y * blockDim.x;
    if (gamma != NULL) {
      int l = 4 * thrx;
      for (; l + 3 < n2; l += 4 * numx) {
        for (int k = 0; k < 4; ++k) {
          const U c_h = static_cast<U>(k_input[l + k]);
          const U c_loss = static_cast<U>(k_dout[l + k]);
          if (!rms_only) {
            sum_loss1 += c_loss * static_cast<U>(gamma[l + k]);
            sum_loss2 += c_loss * static_cast<U>(gamma[l + k]) *
                         (c_h - c_mean) * c_invvar;
          } else {
            sum_loss2 += c_loss * static_cast<U>(gamma[l + k]) * (c_h)*c_invvar;
          }
        }
      }
      for (; l < n2; ++l) {
        const U c_h = static_cast<U>(k_input[l]);
        const U c_loss = static_cast<U>(k_dout[l]);
        if (!rms_only) {
          sum_loss1 += c_loss * static_cast<U>(gamma[l]);
          sum_loss2 +=
              c_loss * static_cast<U>(gamma[l]) * (c_h - c_mean) * c_invvar;
        } else {
          sum_loss2 += c_loss * static_cast<U>(gamma[l]) * (c_h)*c_invvar;
        }
      }
    } else {
      int l = 4 * thrx;
      for (; l + 3 < n2; l += 4 * numx) {
        for (int k = 0; k < 4; ++k) {
          const U c_h = static_cast<U>(k_input[l + k]);
          const U c_loss = static_cast<U>(k_dout[l + k]);
          if (!rms_only) {
            sum_loss1 += c_loss;
            sum_loss2 += c_loss * (c_h - c_mean) * c_invvar;
          } else {
            sum_loss2 += c_loss * (c_h)*c_invvar;
          }
        }
      }
      for (; l < n2; ++l) {
        const U c_h = static_cast<U>(k_input[l]);
        const U c_loss = static_cast<U>(k_dout[l]);
        if (!rms_only) {
          sum_loss1 += c_loss;
          sum_loss2 += c_loss * (c_h - c_mean) * c_invvar;
        } else {
          sum_loss2 += c_loss * (c_h)*c_invvar;
        }
      }
    }
    // intra-warp reductions
    for (int mask = blockDim.x / 2; mask > 0; mask /= 2) {
      if (!rms_only) {
        sum_loss1 += __shfl_xor_sync(0xffffffff, sum_loss1, mask, warpSize);
      }
      sum_loss2 += __shfl_xor_sync(0xffffffff, sum_loss2, mask, warpSize);
    }
    // inter-warp reductions
    if (blockDim.y > 1) {
      SharedMemory<U> shared;
      U *buf = shared.getPointer();
      for (int offset = blockDim.y / 2; offset > 0; offset /= 2) {
        // upper half of warps write to shared
        if (threadIdx.y >= offset && threadIdx.y < 2 * offset) {
          const int wrt_i = (threadIdx.y - offset) * blockDim.x + threadIdx.x;
          if (!rms_only) {
            buf[2 * wrt_i] = sum_loss1;
          }
          buf[2 * wrt_i + 1] = sum_loss2;
        }
        __syncthreads();
        // lower half merges
        if (threadIdx.y < offset) {
          const int read_i = threadIdx.y * blockDim.x + threadIdx.x;
          if (!rms_only) {
            sum_loss1 += buf[2 * read_i];
          }
          sum_loss2 += buf[2 * read_i + 1];
        }
        __syncthreads();
      }
      if (threadIdx.y == 0) {
        if (!rms_only) {
          buf[2 * threadIdx.x] = sum_loss1;
        }
        buf[2 * threadIdx.x + 1] = sum_loss2;
      }
      __syncthreads();
      if (threadIdx.y != 0) {
        if (!rms_only) {
          sum_loss1 = buf[2 * threadIdx.x];
        }
        sum_loss2 = buf[2 * threadIdx.x + 1];
      }
    }
    // all threads now have the two sums over l
    U fH = (U)n2;
    U term1 = (U(1) / fH) * c_invvar;
    T *k_grad_input = grad_input + i1 * n2;
    if (gamma != NULL) {
      for (int l = thrx; l < n2; l += numx) {
        const U c_h = static_cast<U>(k_input[l]);
        const U c_loss = static_cast<U>(k_dout[l]);
        U f_grad_input = fH * c_loss * static_cast<U>(gamma[l]);
        if (!rms_only) {
          f_grad_input -= sum_loss1;
          f_grad_input -= (c_h - c_mean) * c_invvar * sum_loss2;
        } else {
          f_grad_input -= (c_h)*c_invvar * sum_loss2;
        }
        f_grad_input *= term1;
        k_grad_input[l] = static_cast<T>(f_grad_input);
      }
    } else {
      for (int l = thrx; l < n2; l += numx) {
        const U c_h = static_cast<U>(k_input[l]);
        const U c_loss = static_cast<U>(k_dout[l]);
        U f_grad_input = fH * c_loss;
        if (!rms_only) {
          f_grad_input -= sum_loss1;
          f_grad_input -= (c_h - c_mean) * c_invvar * sum_loss2;
        } else {
          f_grad_input -= (c_h)*c_invvar * sum_loss2;
        }
        f_grad_input *= term1;
        k_grad_input[l] = static_cast<T>(f_grad_input);
      }
    }
    // prevent race where buf is written again before reads are done
    __syncthreads();
  }
}

template <typename T, typename U = float, typename V = T>
void HostRMSNormGradient(hipStream_t stream, const V *dout, const U *invvar,
                         const T *input, int n1, int n2, const V *gamma,
                         double epsilon, T *grad_input, V *grad_gamma,
                         int part_size, U *part_grad_gamma) {
  auto getMaxGridY = []() {
    int device;
    int val;
    hipGetDevice(&device);
    hipDeviceGetAttribute(&val, hipDeviceAttributeMaxGridDimY, device);
    return val;
  };
  const uint64_t maxGridY = getMaxGridY();
  if (gamma != NULL) {
    const dim3 threads2(32, 4, 1);
    const dim3 blocks2((n2 + threads2.x - 1) / threads2.x, part_size, 1);
    const int nshared2_a =
        2 * sizeof(U) * threads2.y * threads2.y * (threads2.x + 1);
    const int nshared2_b = threads2.x * threads2.y * sizeof(U);
    const int nshared2 = nshared2_a > nshared2_b ? nshared2_a : nshared2_b;
    // note (mkozuki): I can hard code part_grad_gamma's dtype as float given
    // that the `cuda_layer_norm_gradient` doesn't support double.
    cuComputePartGradGammaBeta<<<blocks2, threads2, nshared2, stream>>>(
        dout, input, n1, n2,
        invvar,                                               // unused
        invvar, U(epsilon), part_grad_gamma, part_grad_gamma, /* unused */
        true);

    const dim3 threads3(32, 8, 1);
    const dim3 blocks3((n2 + threads2.x - 1) / threads2.x, 1, 1);
    const int nshared3 = threads3.x * threads3.y * sizeof(U);
    cuComputeGradGammaBeta<<<blocks3, threads3, nshared3, stream>>>(
        part_grad_gamma, part_grad_gamma,          /* unused */
        part_size, n1, n2, grad_gamma, grad_gamma, /* unused */
        true);
  }

  // compute grad_input
  const dim3 blocks1(1, std::min((uint64_t)n1, maxGridY), 1);
  const dim3 threads1(32, 4, 1);
  int nshared = threads1.y > 1 ? threads1.y * threads1.x * sizeof(U) : 0;
  cuComputeGradInput<<<blocks1, threads1, nshared, stream>>>(
      dout, input, n1, n2, invvar, /* unused */
      invvar, U(epsilon), gamma, grad_input, true);
}

} // namespace

namespace gpu_ops {

void rms_forward_affine_mixed_dtypes(hipStream_t stream, void **buffers,
                                     const char *opaque,
                                     std::size_t opaque_len) {
  const RMSNormDescriptor &d =
      *UnpackDescriptor<RMSNormDescriptor>(opaque, opaque_len);

  DISPATCH_DOUBLE_FLOAT_HALF_AND_BFLOAT_INOUT_TYPES(
      d.x_type, d.w_type, "rms_norm_cuda_kernel",
      HostApplyRMSNorm<scalar_t_in, accscalar_t, scalar_t_out>(
          stream, static_cast<scalar_t_out *>(buffers[2]),
          static_cast<accscalar_t *>(buffers[3]),
          static_cast<scalar_t_in *>(buffers[0]), d.n1, d.n2, d.eps,
          /*gamma=*/static_cast<scalar_t_out *>(buffers[1]));)
}

void rms_backward_affine(hipStream_t stream, void **buffers,
                         const char *opaque, std::size_t opaque_len) {
  const RMSNormDescriptor &d =
      *UnpackDescriptor<RMSNormDescriptor>(opaque, opaque_len);

  DISPATCH_DOUBLE_FLOAT_HALF_AND_BFLOAT_INOUT_TYPES(
      d.x_type, d.w_type, "cuComputeGradInputRMS",
      HostRMSNormGradient(
          stream,
          /*dout=*/static_cast<scalar_t_out *>(buffers[0]),
          /*invvar=*/static_cast<accscalar_t *>(buffers[1]),
          /*input=*/static_cast<scalar_t_in *>(buffers[2]), d.n1, d.n2,
          // TMJ pass NULL argument for gamma, beta, grad_gamma and grad_beta
          // if gamma Tensor is NULL on input.
          /*gamma=*/static_cast<scalar_t_out *>(buffers[3]), d.eps,
          /*grad_input=*/static_cast<scalar_t_in *>(buffers[4]),
          /*grad_gamma=*/static_cast<scalar_t_out *>(buffers[5]),
          d.part_grad_size,
          /*part_grad_gamma=*/static_cast<accscalar_t *>(buffers[6]));)
}

} // namespace gpu_ops
